#include "hip/hip_runtime.h"
/*
 *  dmv_gpu.cu -- Template for DMV GPU kernels
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */ 
#include <stdio.h>
#include "dmv.h"

/*
 *  Utility function to get the thread ID within the
 *  global working space.
 */ 
__device__ int get_global_tid()
{
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Utility function to get the thread ID within the
 *  local/block working space.
 */ 
__device__ int get_local_tid()
{
    return blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Naive kernel
 */ 
__global__ void dmv_gpu_naive(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    /*
     * FILLME: fill the code for the naive kernel.
     */ 
	 size_t tid,j;
	 value_t yi;
	 
	 for (tid=get_global_tid();tid<n;tid+=blockDim.x*gridDim.x)
		{
		yi = 0;
		for (j = 0; j < n; ++j) {
            yi += a[tid*n+j]*x[j];
        }

        y[tid] = yi;
		}
}

/*
 *  Coalesced memory acceses
 */
__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
    /*
     * FILLME: fill the code for the coalesced kernel.
     */ 
	 size_t tid,j;
	 value_t yi;
	 
	 for (tid=get_global_tid();tid<n;tid+=blockDim.x*gridDim.x)
		{
		yi = 0;
		for (j = 0; j < n; ++j) {
            yi += a[j*n+tid]*x[j];
        }

        y[tid] = yi;
		}
}

/*
 *  Use of shared memory
 */
__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    /*
     * FILLME: fill the code for the shared memory kernel.
     */ 
	extern __shared__ value_t x_shared[]; 
	size_t tid,i,j;
	value_t yi;
	
	for (i=get_local_tid();(i<n && i<48*1024/sizeof(value_t));i+=blockDim.x)		x_shared[i]=x[i];
	__syncthreads();
	
	for (tid=get_global_tid();tid<n;tid+=blockDim.x*gridDim.x)
		{
		yi = 0;
		for (j = 0; j < n; ++j) {
			if (j>=48*1024/sizeof(value_t))             yi += a[j*n+tid]*x[j];
			else	yi += a[j*n+tid]*x_shared[j];
        }

        y[tid] = yi;
		}
}
